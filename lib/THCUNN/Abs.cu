#include "hip/hip_runtime.h"
#include "THCUNN.h"
#include "THCHalf.h"
#include "THCHalfAutoNumerics.cuh"
#include <THC/THCApply.cuh>

template <typename T>
struct absupdateOutput_functor
{
  __host__ __device__
  absupdateOutput_functor() = default;

  __device__ void operator()(T* output, const T* input) const
  {
#ifdef __HIP_PLATFORM_HCC__
    *output = fabsf(*input);
#else
    *output = abs(*input);
#endif
  }

  __host__ __device__
  ~absupdateOutput_functor() {}

};

template <typename T>
struct absupdateGradInput_functor
{
  __host__ __device__
  absupdateGradInput_functor() = default;

  __device__ void operator()(T* gradInput, const T* input, const T* gradOutput) const
  {
    *gradInput = *input < 0 ? - *gradOutput : *gradOutput;
  }

  __host__ __device__
  ~absupdateGradInput_functor() {}

};

#include "generic/Abs.cu"
#include "THCGenerateFloatTypes.h"
