#include "hip/hip_runtime.h"
#include "THCUNN.h"
#include "common.h"
#include "THCHalf.h"
#include "THCHalfAutoNumerics.cuh"

#if THRUST_PATH
  #include <thrust/functional.h>
  #include <thrust/device_ptr.h>
  #include <thrust/iterator/zip_iterator.h>
  #include <thrust/transform.h>
  #include <thrust/transform_reduce.h>
#endif

template <typename T>
inline __device__ T eps();

template <>
inline __device__ float eps() { return 1e-12f; }

template <>
inline __device__ double eps() { return 1e-12; }

template <typename Dtype, typename Acctype>
struct bce_functor
{
  template <class Tuple>
  __host__ __device__
  Acctype operator()(Tuple x)
  {
#if THRUST_PATH
    Dtype o = thrust::get<0>(x);
    Dtype t = thrust::get<1>(x);
    return - (t * THCNumerics<Acctype>::log(o + eps<Acctype>()) + (Acctype(1)- t) * THCNumerics<Acctype>::log(Acctype(1) - o + eps<Acctype>()));
#else
    return Acctype(0);
#endif
  }
};

template <typename Dtype, typename Acctype>
struct bce_functor_weights
{
  template <class Tuple>
  __host__ __device__
  Acctype operator()(Tuple x)
  {
#if THRUST_PATH
    Dtype o = thrust::get<0>(x);
    Dtype t = thrust::get<1>(x);
    Dtype w = thrust::get<2>(x);
    return - w * (t * THCNumerics<Acctype>::log(o + eps<Acctype>()) + (Acctype(1) - t) * THCNumerics<Acctype>::log(Acctype(1) - o + eps<Acctype>()));
#else
    return Acctype(0);
#endif
  }
};

template <typename Dtype, typename Acctype>
struct bce_updateGradInput_functor
{
  const Dtype norm;

  bce_updateGradInput_functor(Dtype norm_)
    : norm(norm_)
  {}

  template <class Tuple>
  __host__ __device__
  Dtype operator()(Tuple x)
  {
#if THRUST_PATH
    Dtype o = thrust::get<0>(x);
    Dtype t = thrust::get<1>(x);
    return ScalarConvert<Acctype,Dtype>::to(- (t - o) / ((Acctype(1) - o + eps<Acctype>()) * (o + eps<Acctype>())) * norm);
#else
    return Acctype(0);
#endif
  }
};

template <typename Dtype, typename Acctype>
struct bce_updateGradInput_functor_weights
{
  const Dtype norm;

  bce_updateGradInput_functor_weights(Dtype norm_)
    : norm(norm_)
  {}

  template <class Tuple>
  __host__ __device__
  Dtype operator()(Tuple x)
  {
#if THRUST_PATH
    Dtype o = thrust::get<0>(x);
    Dtype t = thrust::get<1>(x);
    Dtype w = thrust::get<2>(x);
    return ScalarConvert<Acctype, Dtype>::to(- (t - o) / ((Acctype(1) - o + eps<Acctype>()) * (o + eps<Acctype>())) * norm * w);
#else
    return Dtype(0);
#endif
  }
};

#include "generic/BCECriterion.cu"
#include "THCGenerateFloatTypes.h"
