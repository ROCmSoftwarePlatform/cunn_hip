#include "hip/hip_runtime.h"
#include "THCUNN.h"
#include "common.h"
#include "THCHalf.h"
#include "THCHalfAutoNumerics.cuh"

#if THRUST_PATH
  #include <thrust/functional.h>
  #include <thrust/device_ptr.h>
  #include <thrust/iterator/zip_iterator.h>
  #include <thrust/transform.h>
  #include <thrust/transform_reduce.h>
#else
  #include <bolt/amp/iterator/ubiquitous_iterator.h>
  #include <bolt/amp/transform.h>
  #include <bolt/amp/reduce.h>
#endif

template <typename T>
inline __device__ T eps();

template <>
inline __device__ float eps() { return 1e-12f; }

template <>
inline __device__ double eps() { return 1e-12; }

#if THRUST_PATH
#else
  template <typename RealType, typename AccType>
  __global__
  void hipTorch_apply_bce(RealType* in1, 
                          RealType* in2, 
                          AccType* out, 
                          long numElements) 
  {
    CUDA_KERNEL_LOOP(index, numElements) {
      RealType o = in1[index];
      RealType t = in2[index];
      out[index] =
        - (t * THCNumerics<AccType>::log(o + eps<AccType>()) + (AccType(1)- t) * THCNumerics<AccType>::log(AccType(1) - o + eps<AccType>()));
    } 
  }
  
  template <typename RealType, typename AccType>
  __global__
  void hipTorch_apply_bce_weights(RealType* in1, 
                                  RealType* in2, 
                                  RealType* in3, 
                                  AccType* out, 
                                  long numElements) 
  {
    CUDA_KERNEL_LOOP(index, numElements) {
      RealType o = in1[index];
      RealType t = in2[index];
      RealType w = in3[index];
      out[index] =
        - w * (t * THCNumerics<AccType>::log(o + eps<AccType>()) + (AccType(1) - t) * THCNumerics<AccType>::log(AccType(1) - o + eps<AccType>()));
    } 
  }
  
  template <typename RealType, typename AccType>
  __global__
  void hipTorch_apply_updateGradInput(RealType* in1, 
                                      RealType* in2, 
                                      AccType* out, 
                                      long numElements,
                                      RealType norm) 
  {
    CUDA_KERNEL_LOOP(index, numElements) {
      RealType o = in1[index];
      RealType t = in2[index];
      out[index] =
        ScalarConvert<AccType,RealType>::to(- (t - o) / ((AccType(1) - o + eps<AccType>()) * (o + eps<AccType>())) * norm);
    } 
  }
  
  template <typename RealType, typename AccType>
  __global__
  void hipTorch_apply_updateGradInput_weights(RealType* in1, 
                                              RealType* in2, 
                                              RealType* in3, 
                                              AccType* out, 
                                              long numElements,
                                              RealType norm) 
  {
    CUDA_KERNEL_LOOP(index, numElements) {
      RealType o = in1[index];
      RealType t = in2[index];
      RealType w = in3[index];
      out[index] =
        ScalarConvert<AccType, RealType>::to(- (t - o) / ((AccType(1) - o + eps<AccType>()) * (o + eps<AccType>())) * norm * w);
    } 
  }
#endif

template <typename Dtype, typename Acctype>
struct bce_functor
{
#if THRUST_PATH
  template <class Tuple>
  __host__ __device__
  Acctype operator()(Tuple x)
  {
    Dtype o = thrust::get<0>(x);
    Dtype t = thrust::get<1>(x);
    return - (t * THCNumerics<Acctype>::log(o + eps<Acctype>()) + (Acctype(1)- t) * THCNumerics<Acctype>::log(Acctype(1) - o + eps<Acctype>()));
  }
#else
  __host__ __device__
  Acctype operator()(const Dtype& o, const Dtype& t) const
  {
    return - (t * THCNumerics<Acctype>::log(o + eps<Acctype>()) + (Acctype(1)- t) * THCNumerics<Acctype>::log(Acctype(1) - o + eps<Acctype>()));
  }
#endif
};

template <typename Dtype, typename Acctype>
struct bce_functor_weights
{
  template <class Tuple>
  __host__ __device__
  Acctype operator()(Tuple x)
  {
#if THRUST_PATH
    Dtype o = thrust::get<0>(x);
    Dtype t = thrust::get<1>(x);
    Dtype w = thrust::get<2>(x);
    return - w * (t * THCNumerics<Acctype>::log(o + eps<Acctype>()) + (Acctype(1) - t) * THCNumerics<Acctype>::log(Acctype(1) - o + eps<Acctype>()));
#else
    return Acctype(0);
#endif
  }
};

template <typename Dtype, typename Acctype>
struct bce_updateGradInput_functor
{
  const Dtype norm;

  bce_updateGradInput_functor(Dtype norm_)
    : norm(norm_)
  {}

  template <class Tuple>
  __host__ __device__
  Dtype operator()(Tuple x)
  {
#if THRUST_PATH
    Dtype o = thrust::get<0>(x);
    Dtype t = thrust::get<1>(x);
    return ScalarConvert<Acctype,Dtype>::to(- (t - o) / ((Acctype(1) - o + eps<Acctype>()) * (o + eps<Acctype>())) * norm);
#else
    return Acctype(0);
#endif
  }
};

template <typename Dtype, typename Acctype>
struct bce_updateGradInput_functor_weights
{
  const Dtype norm;

  bce_updateGradInput_functor_weights(Dtype norm_)
    : norm(norm_)
  {}

  template <class Tuple>
  __host__ __device__
  Dtype operator()(Tuple x)
  {
#if THRUST_PATH
    Dtype o = thrust::get<0>(x);
    Dtype t = thrust::get<1>(x);
    Dtype w = thrust::get<2>(x);
    return ScalarConvert<Acctype, Dtype>::to(- (t - o) / ((Acctype(1) - o + eps<Acctype>()) * (o + eps<Acctype>())) * norm * w);
#else
    return Dtype(0);
#endif
  }
};

#include "generic/BCECriterion.cu"
#include "THCGenerateFloatTypes.h"
