// WSTHORNTON -- ifdef
#if 1
#include "hip/hip_runtime.h"
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/RReLU.cu"
#else

#include "../common.h"

void THNN_(RReLU_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           THCTensor *noise,
           double lower,
           double upper,
           bool train,
           bool inplace,
           void *generator)
{
  THCUNN_assertSameGPU(state, 3, input, output, noise);
#ifdef CURAND_PATH
  struct hiprandStateMtgp32* gen_states = THCRandom_generatorStates(state);
#else
  struct HipRandStateMtgp32* gen_states = THCRandom_generatorStates(state);
#endif

  if (train)
  {
    input = THCTensor_(newContiguous)(state, input);
    THCTensor_(resizeAs)(state, noise, input);
    real *input_data = THCTensor_(data)(state, input);
    real *noise_data = THCTensor_(data)(state, noise);
    ptrdiff_t n = THCTensor_(nElement)(state, input);
    if (inplace)
    {
#ifdef CURAND_PATH
      hipLaunchKernelGGL((rreluUpdateOutputTrain), dim3(NUM_BLOCKS(n)), dim3(BLOCK_SIZE), 0, THCState_getCurrentStream(state), 
        n, gen_states, input_data, noise_data, input_data, lower, upper);
      THCTensor_(set)(state, output, input);
#else
      hipStream_t currentStream = THCState_getCurrentStream(state);
      hc::accelerator_view* current_accl_view;
      hipHccGetAcceleratorView(currentStream, &current_accl_view);
      // WSTHORNTON (problems for half to float)
      // user_uniform_kernel(*current_accl_view, gen_states, noise_data, user_uniform_functor(lower, upper));
      // hipLaunchKernelGGL((rreluUpdateOutputTrain), dim3(NUM_BLOCKS(n)), dim3(BLOCK_SIZE), 0, THCState_getCurrentStream(state),
      //   n, gen_states, input_data, noise_data, input_data, lower, upper);
      // THCudaTensor_set(state, output, input);
#endif
    }
    else
    {
      THCTensor_(resizeAs)(state, output, input);
      real *output_data = THCTensor_(data)(state, output);
#ifdef CURAND_PATH
      hipLaunchKernelGGL((rreluUpdateOutputTrain), dim3(NUM_BLOCKS(n)), dim3(BLOCK_SIZE), 0, THCState_getCurrentStream(state), 
        n, gen_states, input_data, noise_data, output_data, lower, upper);
#else
      hipStream_t currentStream = THCState_getCurrentStream(state);
      hc::accelerator_view* current_accl_view;
      hipHccGetAcceleratorView(currentStream, &current_accl_view);
      // WSTHORNTON (problems for half to float)
      // user_uniform_kernel(*current_accl_view, gen_states, noise_data, user_uniform_functor(lower, upper));
      // hipLaunchKernelGGL((rreluUpdateOutputTrain), dim3(NUM_BLOCKS(n)), dim3(BLOCK_SIZE), 0, THCState_getCurrentStream(state),
      //   n, gen_states, input_data, noise_data, output_data, lower, upper);
#endif
    }
    THCudaCheck(hipGetLastError());
    THCTensor_(free)(state, input);
  }
  else
  {
    const real negSlope = ScalarConvert<double, real>::to((lower + upper) / 2);
    if (inplace)
    {
      THC_pointwiseApply1(state, input, RReLUUpdateOutputEvalIP_functor<real>(negSlope));
      THCTensor_(set)(state, output, input);
    }
    else
    {
      THCTensor_(resizeAs)(state, output, input);
      THC_pointwiseApply2(state, output, input, RReLUUpdateOutputEval_functor<real>(negSlope));
    }
  }
}

void THNN_(RReLU_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           THCTensor *noise,
           double lower,
           double upper,
           bool train,
           bool inplace)
{
  THCUNN_check_nElement(state, input, gradOutput);
  THCUNN_assertSameGPU(state, 4, input, gradOutput, gradInput, noise);

  gradOutput = THCTensor_(newContiguous)(state, gradOutput);

  if (train && upper - lower > 1E-6)    // e.g. if upper == lower, RReLU behaves like LeakyReLU
  {
    // multiply the gradient by the noise tensor
    if (inplace)
    {
      THCTensor_(cmul)(state, gradOutput, gradOutput, noise);
      THCTensor_(set)(state, gradInput, gradOutput);
    }
    else
    {
      THCTensor_(resizeAs)(state, gradInput, input);
      THCTensor_(cmul)(state, gradInput, gradOutput, noise);
    }
  }
  else
  {
    // use constant factor for negative input values
    const real negSlope = ScalarConvert<double, real>::to((lower + upper) / 2);
    if (inplace)
    {
      THC_pointwiseApply2(state, gradOutput, input, RReLUupdateGradInputEvalIP_functor<real>(negSlope));
      THCTensor_(set)(state, gradInput, gradOutput);
    }
    else
    {
      THCTensor_(resizeAs)(state, gradInput, input);
      THC_pointwiseApply3(state, gradInput, gradOutput, input, RReLUupdateGradInputEval_functor<real>(negSlope));
    }
  }

  THCTensor_(free)(state, gradOutput);
}

#endif
#endif
