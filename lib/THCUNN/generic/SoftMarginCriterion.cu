
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/SoftMarginCriterion.cu"
#else

void THNN_(SoftMarginCriterion_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *target,
           THCTensor *output,
           bool sizeAverage)
{
  THCUNN_check_nElement(state, input, target);
  THCUNN_check_dim_size(state, output, 1, 0, 1);
  THCUNN_assertSameGPU(state, 2, input, target);
  accreal sum;

  ptrdiff_t size = THCTensor_(nElement)(state, input);

  input = THCTensor_(newContiguous)(state, input);
  target = THCTensor_(newContiguous)(state, target);

#if THRUST_PATH
  thrust::device_ptr<real> input_data(THCTensor_(data)(state, input));
  thrust::device_ptr<real> target_data(THCTensor_(data)(state, target));
  sum = thrust::inner_product(input_data, input_data+size, target_data, (accreal) 0, thrust::plus<accreal>(), softmargin_functor<real, accreal>());
#else
  auto input_data = make_ubiquitous_iterator(THCTensor_(data)(state, input));
  auto target_data = make_ubiquitous_iterator(THCTensor_(data)(state, target));
  sum = bolt::amp::inner_product(input_data, input_data+size, target_data, (accreal) 0, bolt::amp::plus<accreal>(), softmargin_functor<real, accreal>());
#endif

  if(sizeAverage)
    sum /= size;

  THCTensor_(free)(state, input);
  THCTensor_(free)(state, target);

  THCTensor_(set1d)(state, output, 0, ScalarConvert<accreal, real>::to(sum));
}

void THNN_(SoftMarginCriterion_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *target,
           THCTensor *gradInput,
           bool sizeAverage)
{
  THCUNN_check_nElement(state, input, target);
  THCUNN_assertSameGPU(state, 3, input, target, gradInput);

  ptrdiff_t size = THCTensor_(nElement)(state, input);
  accreal norm = (sizeAverage ? 1./size : 1.);

  input = THCTensor_(newContiguous)(state, input);
  target = THCTensor_(newContiguous)(state, target);

  THCTensor_(resizeAs)(state, gradInput, input);

#if THRUST_PATH
  thrust::device_ptr<real> input_data(THCTensor_(data)(state, input));
  thrust::device_ptr<real> target_data(THCTensor_(data)(state, target));
  thrust::device_ptr<real> gradInput_data(THCTensor_(data)(state, gradInput));

  thrust::transform(input_data, input_data+size, target_data, gradInput_data, softmargin_updateGradInput_functor<real, accreal>(norm));
#else
  auto input_data = make_ubiquitous_iterator(THCTensor_(data)(state, input));
  auto target_data = make_ubiquitous_iterator(THCTensor_(data)(state, target));
  auto gradInput_data = make_ubiquitous_iterator(THCTensor_(data)(state, gradInput));

  bolt::amp::transform(input_data, input_data+size, target_data, gradInput_data, softmargin_updateGradInput_functor<real, accreal>(norm));
#endif
  THCTensor_(free)(state, input);
  THCTensor_(free)(state, target);
}

#endif
