#include "THCUNN.h"
#include "THCHalf.h"
#include "THCHalfAutoNumerics.cuh"

#ifdef __NVCC__
  #include <hipsparse.h>
#endif

#ifdef __NVCC__
static hipsparseHandle_t cusparse_handle = 0;
#endif

static void init_cusparse() {
#ifdef __NVCC__
  if (cusparse_handle == 0) {
    hipsparseStatus_t status = hipsparseCreate(&cusparse_handle);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
      THError("CUSPARSE Library initialization failed");
    }
  }
#endif
}

#ifdef CUDA_HALF_TENSOR
void THNN_CudaHalfSparseLinear_updateOutput(
          THCState *state,
          THCudaHalfTensor *input,
          THCudaHalfTensor *output,
          THCudaHalfTensor *weight,
          THCudaHalfTensor *bias) {
  THError("THCudaHalfTensor not supported with SparseLinear");
}

void THNN_CudaHalfSparseLinear_accGradParameters(
          THCState *state,
          THCudaHalfTensor *input,
          THCudaHalfTensor *gradOutput,
          THCudaHalfTensor *gradWeight,
          THCudaHalfTensor *gradBias,
          THCudaHalfTensor *weight,
          THCudaHalfTensor *bias,
          float weightDecay,
          float scale) {
  THError("THCudaHalfTensor not supported with SparseLinear");
}

void THNN_CudaHalfSparseLinear_legacyUpdateOutput(
          THCState *state,
          THCudaHalfTensor *input,
          THCudaHalfTensor *output,
          THCudaHalfTensor *weight,
          THCudaHalfTensor *bias) {
  THError("THCudaHalfTensor not supported with SparseLinear");
}

void THNN_CudaHalfSparseLinear_legacyAccGradParameters(
          THCState *state,
          THCudaHalfTensor *input,
          THCudaHalfTensor *gradOutput,
          THCudaHalfTensor *gradWeight,
          THCudaHalfTensor *gradBias,
          THCudaHalfTensor *weight,
          THCudaHalfTensor *bias,
          float weightDecay,
          float scale) {
  THError("THCudaHalfTensor not supported with SparseLinear");
}

void THNN_CudaHalfSparseLinear_zeroGradParameters(
          THCState *state,
          THCudaHalfTensor *gradWeight,
          THCudaHalfTensor *gradBias,
          THCudaHalfTensor *lastInput) {
  THError("THCudaHalfTensor not supported with SparseLinear");
}

void THNN_CudaHalfSparseLinear_updateParameters(
          THCState *state,
          THCudaHalfTensor *weight,
          THCudaHalfTensor *bias,
          THCudaHalfTensor *gradWeight,
          THCudaHalfTensor *gradBias,
          THCudaHalfTensor *lastInput,
          float learningRate) {
  THError("THCudaHalfTensor not supported with SparseLinear");
}
#endif

#include "generic/SparseLinear.cu"
#include "THCGenerateFloatType.h"
#include "generic/SparseLinear.cu"
#include "THCGenerateDoubleType.h"
