#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "THCUNN.h"
#include "THCHalf.h"
#include "THCHalfAutoNumerics.cuh"

#define SOFTMAX_THREADS 128

template <typename T, typename AccumT>
__global__ void cunn_SoftMax_updateOutput_kernel(
  T *output, T *input, int nframe, int dim, int stride0, int stride1)
{
  __shared__ AccumT buffer[SOFTMAX_THREADS+1];
  T *input_k  = input  + hipBlockIdx_x*dim*stride0 + hipBlockIdx_y*stride1 + hipBlockIdx_z;
  T *output_k = output + hipBlockIdx_x*dim*stride0 + hipBlockIdx_y*stride1 + hipBlockIdx_z;

  int i_start = hipThreadIdx_x;
  int i_end = dim;
  int i_step = hipBlockDim_x;

  // max?
  buffer[hipThreadIdx_x] = -THCNumerics<AccumT>::max();
  for (int i=i_start; i<i_end; i+=i_step)
  {
    T z = input_k[i*stride0];
    AccumT zAcc = ScalarConvert<T, AccumT>::to(z);
    if (buffer[hipThreadIdx_x] < zAcc)
      buffer[hipThreadIdx_x] = zAcc;
  }


  __syncthreads();

  // reduce
  if (hipThreadIdx_x == 0)
  {
    AccumT max_k = -THCNumerics<AccumT>::max();
    for (int i=0; i<hipBlockDim_x; i++)
    {
      if (max_k < buffer[i])
        max_k = buffer[i];
    }
    buffer[SOFTMAX_THREADS] = max_k;
  }

  __syncthreads();

  // sum?
  T max_k = ScalarConvert<AccumT, T>::to(buffer[SOFTMAX_THREADS]);
  buffer[hipThreadIdx_x] = ScalarConvert<int, AccumT>::to(0);
  for (int i=i_start; i<i_end; i+=i_step) {
    T z = THCNumerics<T>::exp(input_k[i*stride0]-max_k);
    buffer[hipThreadIdx_x] += ScalarConvert<T, AccumT>::to(z);
    output_k[i*stride0] = z;
  }

  __syncthreads();

  // reduce
  if (hipThreadIdx_x == 0)
  {
    AccumT sum_k = ScalarConvert<int, AccumT>::to(0);
    for (int i=0; i<hipBlockDim_x; i++)
      sum_k += buffer[i];
    buffer[SOFTMAX_THREADS] = sum_k;
  }

  __syncthreads();

  // softmax
  T sum_k = ScalarConvert<AccumT, T>::to(buffer[SOFTMAX_THREADS]);
  for (int i=i_start; i<i_end; i+=i_step)
    output_k[i*stride0] = output_k[i*stride0] / sum_k;
}

template <typename T, typename AccumT>
__global__ void cunn_SoftMax_updateGradInput_kernel(
  T *gradInput, T *output, T *gradOutput, int nframe, int dim, int stride0, int stride1)
{
  __shared__ AccumT buffer[SOFTMAX_THREADS];
  T *gradInput_k  = gradInput  + hipBlockIdx_x*dim*stride0 + hipBlockIdx_y * stride1 + hipBlockIdx_z;
  T *output_k     = output     + hipBlockIdx_x*dim*stride0 + hipBlockIdx_y * stride1 + hipBlockIdx_z;
  T *gradOutput_k = gradOutput + hipBlockIdx_x*dim*stride0 + hipBlockIdx_y * stride1 + hipBlockIdx_z;

  int i_start = hipThreadIdx_x;
  int i_end = dim;
  int i_step = hipBlockDim_x;

  // sum?
  buffer[hipThreadIdx_x] = ScalarConvert<int, AccumT>::to(0);
  for (int i=i_start; i<i_end; i+=i_step)
    buffer[hipThreadIdx_x] += ScalarConvert<T, AccumT>::to(gradOutput_k[i*stride0] * output_k[i*stride0]);

  __syncthreads();

  // reduce
  if (hipThreadIdx_x == 0)
  {
    AccumT sum_k = ScalarConvert<int, AccumT>::to(0);
    for (int i=0; i<hipBlockDim_x; i++)
      sum_k += buffer[i];
    buffer[0] = sum_k;
  }

  __syncthreads();

  T sum_k = ScalarConvert<AccumT, T>::to(buffer[0]);
  for (int i=i_start; i<i_end; i+=i_step)
    gradInput_k[i*stride0] = output_k[i*stride0] * (gradOutput_k[i*stride0] - sum_k);
}

#include "generic/SoftMax.cu"
#include "THCGenerateFloatTypes.h"

#undef SOFTMAX_THREADS
